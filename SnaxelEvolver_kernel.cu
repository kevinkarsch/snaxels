#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, kernels
#include <hip/hip_runtime.h>
#include <cutil_math.h>
#include <cutil_inline.h>

__global__ void findInitVertsKernel(float3 *vertPositions_d,
									int numVerts,
									int *nValenceVerts_d,
									int *idxValenceVerts_d,
									float3 *valenceEdgeNormals_d,
									float3 viewpoint,
									bool *initIndices_d)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx >= numVerts || !initIndices_d[idx]) return;
	float3 vv = normalize(vertPositions_d[idx] - viewpoint);
	//Go through two edges at a time
	int size = nValenceVerts_d[idx]-1;
	for(int i = 0; i < size; i++) {
		// Check if the view vector lies inside of these normals on the gauss map sphere
		float3 polyNormal = normalize( cross(valenceEdgeNormals_d[i+idxValenceVerts_d[idx]],
								             valenceEdgeNormals_d[i+1+idxValenceVerts_d[idx]]) );
		if(dot(vv,valenceEdgeNormals_d[i+idxValenceVerts_d[idx]]) > 0 || dot(vv, polyNormal) > 0) {
			initIndices_d[idx] = false;
			return;
		}
	}
}

extern "C" void findInitVerts(float3 *vertPositions_d,
								int numVerts,
								int *nValenceVerts_d,
								int *idxValenceVerts_d,
								float3 *valenceEdgeNormals_d,
								float3 viewpoint,
								bool *initIndices_d,
								bool *initIndices) 
{
	//Set kernel params
	int blocksPerGrid = 1024;
	int threadsPerBlock = int(ceil(float(numVerts)/float(blocksPerGrid)));
	dim3 grids(blocksPerGrid, 1, 1);
	dim3 blocks(threadsPerBlock, 1, 1);
	//Run kernel
	findInitVertsKernel <<< grids, blocks >>> (vertPositions_d, numVerts, 
			nValenceVerts_d, idxValenceVerts_d, valenceEdgeNormals_d, 
			viewpoint, initIndices_d);
	//cudaThreadsSyncronize();
	//Copy device memory back to host
    cutilSafeCall(hipMemcpy(initIndices, initIndices_d, numVerts*sizeof(bool), hipMemcpyDeviceToHost));
}
